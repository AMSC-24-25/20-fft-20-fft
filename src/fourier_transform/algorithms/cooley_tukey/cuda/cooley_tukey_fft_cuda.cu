#include "hip/hip_runtime.h"
#include <vector>
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdexcept>
#include "cooley_tukey_fft_cuda.hpp"

namespace fft::algorithms::cooley_tukey
{
    __device__ hipDoubleComplex W(int N, int k) {
        double angle = -2.0 * M_PI * k / N;
        return make_hipDoubleComplex(cos(angle), sin(angle));
    }

    __global__ void conjugate_and_scale(hipDoubleComplex* data, int N, bool scale) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid < N) {
            data[tid] = hipConj(data[tid]);
            if (scale) {
                data[tid].x /= N;
                data[tid].y /= N;
            }
        }
    }

    __global__ void fft_kernel(hipDoubleComplex* data, int N, int logN) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;

        for (int s = 1; s <= logN; ++s) {
            int m = 1 << s;
            int m2 = m >> 1;

            if (tid < N / 2) {
                int j = tid % m2;
                int k = tid - j + (tid / m2) * m2;

                hipDoubleComplex t = hipCmul(W(m, j), data[k + m2]);
                hipDoubleComplex u = data[k];
                data[k]       = hipCadd(u, t);
                data[k + m2]  = hipCsub(u, t);
            }

            __syncthreads();
        }
    }

    void computeFFTCuda(std::vector<std::complex<double>>& input) {
        const int N = input.size();

        int logN = static_cast<int>(std::log2(N));

        hipDoubleComplex* d_data;
        hipMalloc(&d_data, N * sizeof(hipDoubleComplex));

        std::vector<hipDoubleComplex> tmp(N);
        for (int i = 0; i < N; ++i)
            tmp[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());

        hipMemcpy(d_data, tmp.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

        int threads = 256;
        int blocks = (N + threads - 1) / threads;
        fft_kernel<<<blocks, threads>>>(d_data, N, logN);
        hipDeviceSynchronize();

        hipMemcpy(tmp.data(), d_data, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        for (int i = 0; i < N; ++i)
            input[i] = std::complex<double>(hipCreal(tmp[i]), hipCimag(tmp[i]));

        hipFree(d_data);
    }

    void computeInverseFFTCuda(std::vector<std::complex<double>>& input) {
        const int N = input.size();

        int logN = static_cast<int>(std::log2(N));

        hipDoubleComplex* d_data;
        hipMalloc(&d_data, N * sizeof(hipDoubleComplex));

        std::vector<hipDoubleComplex> tmp(N);
        for (int i = 0; i < N; ++i)
            tmp[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());

        hipMemcpy(d_data, tmp.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

        int threads = 256;
        int blocks = (N + threads - 1) / threads;

        // Conjugate input
        conjugate_and_scale<<<blocks, threads>>>(d_data, N, false);

        // Forward FFT
        fft_kernel<<<blocks, threads>>>(d_data, N, logN);
        hipDeviceSynchronize();

        // Conjugate again and scale by 1/N
        conjugate_and_scale<<<blocks, threads>>>(d_data, N, true);
        hipDeviceSynchronize();

        hipMemcpy(tmp.data(), d_data, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        for (int i = 0; i < N; ++i)
            input[i] = std::complex<double>(hipCreal(tmp[i]), hipCimag(tmp[i]));

        hipFree(d_data);
    }
}
